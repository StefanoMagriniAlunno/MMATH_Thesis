#include "hip/hip_runtime.h"
/**
 * @file cudafcm.cu
 * @author Stefano Magrini Alunno (stefanomagrini99@gmail.com)
 * @brief definition of cudafcm function
 *
 * @date 2024-07-20
 *
 * @copyright Copyright (c) 2024
 */
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <thread>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform_reduce.h>

#include <hipblas.h>

#include <omp.h>

#include "fcm.h"

// reserved memory for the system in GiB (a float castable value)
#define GiB_SYS_RESERVED_MEM 1.0
// max number of threads per block
#define MAX_THREADS_PER_BLOCK 128

// used to pass a log instruction
#define LOGGER(log_stream, level, message)                                    \
  do                                                                          \
    {                                                                         \
      log_stream << __FILE__ << ":" << __LINE__ << " \t" << (level) << " "    \
                 << (message) << std::endl;                                   \
      log_stream.flush ();                                                    \
    }                                                                         \
  while (0)
// check for errors in the runtime
#define CHECK_ERROR_RUNTIME_ERROR(assertion, closing_instruction, message)    \
  if (!(assertion))                                                           \
    {                                                                         \
      LOGGER (log_stream, "ERROR caught", message);                           \
      {                                                                       \
        closing_instruction;                                                  \
      }                                                                       \
      throw std::runtime_error (message);                                     \
    }
// check for errors for bad allocation
#define CHECK_ERROR_BAD_ALLOC(assertion, closing_instruction, message)        \
  if (!(assertion))                                                           \
    {                                                                         \
      LOGGER (log_stream, "ERROR caught", message);                           \
      log_stream.flush ();                                                    \
      {                                                                       \
        closing_instruction;                                                  \
      }                                                                       \
      throw std::bad_alloc ();                                                \
    }
// used to pass an empty instruction
#define PASS_INSTRUCTION ;

/// @cond
float
tocputool (const float *const d_addr)
{
  float value;
  hipMemcpy (&value, d_addr, sizeof (float), hipMemcpyDeviceToHost);
  return value;
}
/// @endcond

/**
 * @brief struct used to manage the memory partitions in the device
 * In this way the memory is allocated only once and the data is
 * stored in a contiguous way.
 *
 */
struct partition
{
  float *d_new_centroids; /*!< device memory for new centroids */
  float *d_centroids;     /*!< device memory for centroids */
  float *d_data;       /*!< device memory for data points in a single batch */
  float *d_weights;    /*!< device memory for weights of data points */
  float *d_matrix;     /*!< device memory for matrix of distances */
  float *d_energies;   /*!< device memory for energies of data points */
  size_t batch_size;   /*!< number of data points to use in a single
                          batch */
  size_t n_centroids;  /*!< number of centroids */
  size_t n_dimensions; /*!< number of dimensions */
};

/**
 * @brief used in update_centroids to access data in a non-contiguous
 * way This struct is used to access data in a non-contiguous way
 *
 */
struct non_contiguous_access
{
  float *data;   /*!< address of the data */
  size_t stride; /*!< stride of the data, n_dimensions */

  /*!< constructor */
  __host__ __device__
  non_contiguous_access (float *data, size_t stride)
      : data (data), stride (stride)
  {
  }

  /*!< operator */
  __host__ __device__ float
  operator() (size_t i) const
  {
    return data[i * stride];
  }
};

/**
 * @brief This kernel computes the matrix U2 of membership between
 * data points and centroids
 *
 * @param[in] d_data : the i-th is d_data[i * n_dimensions + k]
 * for k = 0, ..., n_dimensions - 1
 * @param[in] d_weights : the weight of the i-th data point is
 * d_weights[i]
 * @param[in] d_centroids : the j-th is
 * d_centroids[j * n_dimensions + k] for k = 0, ..., n_dimensions - 1
 * @param[out] d_matrix : the weighted membership between the i-th data point
 * and the j-th centroid is stored in d_matrix[i * n_centroids + j]
 * @param[out] d_energies : the energy of the i-th data point is stored in
 * d_energies[i]
 * @param n_data : number of data points
 * @param n_dimensions : dimensions of data points
 * @param n_centroids : number of centroids
 *
 * @details This kernel requires a grid of blocks with n_data blocks
 * and MAX_THREADS_PER_BLOCK threads for each block.
 *
 * @note This kernel synchronize threads at the end of the computation
 */
__global__ void
kernel_compute_U2 (const float *const d_data, const float *const d_weights,
                   const float *const d_centroids, float *const d_matrix,
                   float *const d_energies, size_t n_data, size_t n_dimensions,
                   size_t n_centroids)
{
  __shared__ float sdata[MAX_THREADS_PER_BLOCK];
  size_t i = blockIdx.x;  // i-th data
  size_t j = threadIdx.x; // j-th centroid
  float value = 0;
  float min_value = 0;
  float d2 = 0;

  // compute the distance between the i-th data point and the j-th
  // centroid
  if (i < n_data && j < n_centroids)
    {
      for (size_t k = 0; k < n_dimensions; k++)
        {
          float diff = d_data[i * n_dimensions + k]
                       - d_centroids[j * n_dimensions + k];
          value += diff * diff;
        }
    }
  d2 = value;
  // syncronyze threads of this block
  __syncthreads ();

  // compute the min value of the block
  if (j < n_centroids)
    sdata[j] = value;
  else
    sdata[j] = FLT_MAX;
  __syncthreads ();
  for (size_t s = MAX_THREADS_PER_BLOCK / 2; s > 0; s >>= 1)
    {
      if (j < s && sdata[j] > sdata[j + s])
        sdata[j] = sdata[j + s];
      __syncthreads ();
    }
  min_value = sdata[0];
  // syncronyze threads of this block
  __syncthreads ();

  // prepare the row to a stable normalization
  if (min_value == 0.0)
    {
      // let to 1 the components that are 0 and to 0 the others
      if (i < n_data && j < n_centroids)
        value = value == 0.0 ? 1.0 : 0.0;
    }
  else
    {
      // for each component of the row, assign min/value
      if (i < n_data && j < n_centroids)
        value = min_value / value;
    }
  // syncronyze threads of this block
  __syncthreads ();

  // compute the sum of the row
  if (j < n_centroids)
    sdata[j] = value;
  else
    sdata[j] = 0.0;
  __syncthreads ();
  for (size_t s = MAX_THREADS_PER_BLOCK / 2; s > 0; s >>= 1)
    {
      if (j < s)
        sdata[j] += sdata[j + s];
      __syncthreads ();
    }
  min_value = sdata[0];
  // syncronyze threads of this block
  __syncthreads ();

  // assign the value to the matrix
  if (i < n_data && j < n_centroids) {
    value /= min_value;
    d_matrix[i * n_centroids + j] = value * value * d_weights[i];
  }
  // syncronyze threads of this block
  __syncthreads ();

  // compute energy
  if (i < n_data && j < n_centroids)
    value = d_matrix[i * n_centroids + j] * d2;  // compute partial energy
  // syncronyze threads of this block
  __syncthreads ();

  // compute the sum of the partial energies
  if (j < n_centroids)
    sdata[j] = value;
  else
    sdata[j] = 0.0;
  __syncthreads ();
  for (size_t s = MAX_THREADS_PER_BLOCK / 2; s > 0; s >>= 1)
    {
      if (j < s)
        sdata[j] += sdata[j + s];
      __syncthreads ();
    }
  value = sdata[0];  // energy of the data point
  // syncronyze threads of this block
  __syncthreads ();

  // assign the energy to the matrix
  if (i < n_data && j == 0)
    d_energies[i] = value;
  // syncronyze threads of this block
  __syncthreads ();
}

/**
 * @brief Update centroids yet allocated on the device
 *
 * @param[in] d_data : data points stored as d_data[i * n_dimensions +
 * k] for k = 0, ..., n_dimensions - 1
 * @param[in] d_matrix : matrix of distances stored as d_matrix[i *
 * n_centroids
 * + j]
 * @param[out] h_centroids_weight : weights of centroids
 * @param[out] d_new_centroids : new centroids
 * @param n_data : number of data points
 * @param n_dimensions : dimensions of data points
 * @param n_centroids : number of centroids
 * @param prop : properties of the device
 * @param handle : cublas handle
 * @param log_stream : log file
 *
 * @exception std::runtime_error : if an error occurs during the
 * computation
 *
 * @note This function synchronize threads at the end of the
 * computation
 */
__host__ void
update_centroids (const float *const d_data, float *const d_matrix,
                  float *const h_centroids_weight,
                  float *const d_new_centroids, size_t n_data,
                  size_t n_dimensions, size_t n_centroids,
                  const hipDeviceProp_t &prop, hipblasHandle_t handle,
                  std::ofstream &log_stream)
{
  // use cublas to compute the new centroids
  hipblasStatus_t status;
  hipError_t err;

  // compute the product of d_matrix and d_data
  float alpha = 1.0;
  float beta = 1.0;
  // for  k=1:n_dimensions, j=1:n_centroids
  // d_new_centroids[k + j*n_dimensions] = d_new_centroids[k +
  // j*n_dimensions]
  // + sum_i=1:n_data d_data[k + i*n_dimensions]*d_matrix[j +
  // i*n_centroids]
  status = hipblasSgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n_dimensions,
                        n_centroids, n_data, &alpha, d_data, n_dimensions,
                        d_matrix, n_centroids, &beta, d_new_centroids,
                        n_dimensions);

  // check error
  CHECK_ERROR_RUNTIME_ERROR (status == HIPBLAS_STATUS_SUCCESS, PASS_INSTRUCTION,
                             "CUBLAS sgemm failed with status "
                                 + std::to_string (status));

  /// @remark it's not necessary to syncronize threads

  for (size_t i = 0; i < n_centroids; i++)
    {
      // compute the sum of d_matrix along data points
      try
        {
          float *d_matrix_ptr = thrust::raw_pointer_cast (d_matrix + i);
          auto begin = thrust::make_transform_iterator (
              thrust::counting_iterator<size_t> (0),
              non_contiguous_access (d_matrix_ptr, n_centroids));
          auto end = begin + n_data;
          h_centroids_weight[i] += thrust::transform_reduce (
              begin, end, thrust::identity<float> (), 0.0f,
              thrust::plus<float> ());
        }
      catch (std::runtime_error &e)
        {
          LOGGER (log_stream, "CRITICAL caught", e.what ());
          throw std::runtime_error (e.what ());
        }
    }

  // syncronize threads
  err = hipDeviceSynchronize ();
  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, PASS_INSTRUCTION,
                             hipGetErrorString (err));
}

/**
 * @brief This function compute the matrix of distances between data
 * points and centroids
 *
 * @param[in] d_data : data points stored as d_data[i * n_dimensions +
 * k] for k = 0, ..., n_dimensions - 1
 * @param[in] d_weights : weights of data points
 * @param[in] d_centroids : centroids stored as d_centroids[j *
 * n_dimensions + k] for k = 0, ..., n_dimensions - 1
 * @param[out] d_matrix : matrix of distances stored as d_matrix[i *
 * n_centroids + j] for j = 0, ..., n_centroids - 1
 * @param n_data : number of data points
 * @param n_dimensions : dimensions of data points
 * @param n_centroids : number of centroids
 * @param prop : properties of the device
 * @param log_stream : log file
 *
 * @exception std::runtime_error : if an error occurs during the
 * computation
 *
 * @note This function synchronize threads at the end of the
 * computation
 */
__host__ void
compute_U2 (const float *const d_data, const float *const d_weights,
            const float *const d_centroids, float *const d_matrix,
            float *const d_energies, size_t n_data, size_t n_dimensions,
            size_t n_centroids, const hipDeviceProp_t &prop, std::ofstream &log_stream)
{
  hipError_t err;

  // each block works on a single data point
  dim3 grid (n_data);
  dim3 block (MAX_THREADS_PER_BLOCK);

  // call the kernel
  // clang-format off
  kernel_compute_U2<<<grid, block>>> (d_data, d_weights, d_centroids, d_matrix, d_energies, n_data,
                                        n_dimensions, n_centroids);
  // clang-format on

  // check for errors
  err = hipGetLastError ();
  CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, PASS_INSTRUCTION,
                             hipGetErrorString (err));

  // synchronize threads
  err = hipDeviceSynchronize ();
  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, PASS_INSTRUCTION,
                             hipGetErrorString (err));
}

/**
 * @brief This function compute centroids
 *
 * @param data : data points stored as data[i * n_dimensions + k] for
 * k = 0,
 * ..., n_dimensions - 1
 * @param weights : weights of data points
 * @param partitions : partitions of the data points
 * @param prop : properties of the device
 * @param log_stream : log file
 * @return float : energy of the data points
 *
 * @details For a better performance, the log messages are written
 * only in case of error.
 *
 * @exception std::runtime_error : if an error occurs during the
 * computation
 * @exception std::bad_alloc : if an error occurs during the memory
 * allocation
 */
__host__ float
compute_centroids (const std::vector<float> &data,
                   const std::vector<float> &weights,
                   const struct partition partitions,
                   const hipDeviceProp_t &prop, std::ofstream &log_stream)
{
  // cicle over the batches
  hipError_t err;
  hipblasStatus_t status;
  hipblasHandle_t handle;

  // initialize cublas for U2 computation
  status = hipblasCreate (&handle);
  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (status == HIPBLAS_STATUS_SUCCESS, PASS_INSTRUCTION,
                             "CUBLAS initialization failed with status "
                                 + std::to_string (status));

  // d_new_centroids is a zero-initialized vector
  err = hipMemset (partitions.d_new_centroids, 0,
                    partitions.n_centroids * partitions.n_dimensions
                        * sizeof (float));
  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, hipblasDestroy (handle),
                             hipGetErrorString (err));

  // allocate memory for the centroids weight
  float *h_centroids_weight
      = (float *)calloc (partitions.n_centroids, sizeof (float));
  // check for errors
  CHECK_ERROR_BAD_ALLOC (h_centroids_weight != NULL, hipblasDestroy (handle),
                         "Memory allocation failed");

  // define energy of the data points
  float energy = 0;

  // cicle over the data points
  size_t n_data = data.size () / partitions.n_dimensions;
  size_t c_data = 0; // completed data points
  while (c_data < n_data)
    {
      size_t batch_size = std::min (partitions.batch_size, n_data - c_data);

      // report completion
      LOGGER (log_stream, "INFO",
              "completed " + std::to_string (c_data) + " ("
                  + std::to_string (100 * (double)c_data / (double)n_data)
                  + "%) data points");

      // copy data to the device
      err = hipMemcpy (partitions.d_data,
                        data.data () + c_data * partitions.n_dimensions,
                        batch_size * partitions.n_dimensions * sizeof (float),
                        hipMemcpyHostToDevice);
      // check for errors
      CHECK_ERROR_RUNTIME_ERROR (
          err == hipSuccess,
          {
            hipblasDestroy (handle);
            free (h_centroids_weight);
          },
          hipGetErrorString (err));

      // copy weights to the device
      err = hipMemcpy (partitions.d_weights, weights.data () + c_data,
                        batch_size * sizeof (float), hipMemcpyHostToDevice);
      // check for errors
      CHECK_ERROR_RUNTIME_ERROR (
          err == hipSuccess,
          {
            hipblasDestroy (handle);
            free (h_centroids_weight);
          },
          hipGetErrorString (err));

      try
        {
          // compute the matrix U2
          compute_U2 (partitions.d_data, partitions.d_weights,
                      partitions.d_centroids, partitions.d_matrix, partitions.d_energies,
                      batch_size, partitions.n_dimensions, partitions.n_centroids, prop,
                      log_stream);

          // update the new centroids
          update_centroids (partitions.d_data, partitions.d_matrix,
                            h_centroids_weight, partitions.d_new_centroids,
                            batch_size, partitions.n_dimensions,
                            partitions.n_centroids, prop, handle, log_stream);
          // compute batch energy as the sum of the energies
          // use thrust::transform_reduce
          float *batch_energy_ptr = thrust::raw_pointer_cast (partitions.d_energies);
          auto begin = thrust::make_transform_iterator (
              thrust::counting_iterator<size_t> (0),
              non_contiguous_access (batch_energy_ptr, 1));
          auto end = begin + batch_size;
          float batch_energy = thrust::transform_reduce (
              begin, end, thrust::identity<float> (), 0.0f, thrust::plus<float> ());
          // update the total energy
          energy += batch_energy;
        }
      catch (std::runtime_error &e)
        {
          LOGGER (log_stream, "CRITICAL caught", e.what ());
          {
            hipblasDestroy (handle);
            free (h_centroids_weight);
          }
          throw std::runtime_error (e.what ());
        }

      c_data += batch_size;
    }

  // divide d_new_centroids by the centroids weight
  for (size_t i = 0; i < partitions.n_centroids; i++)
    {
      if (h_centroids_weight[i] != 0)
        {
          // use cublas to divide d_new_centroids[i,:] by
          // h_centroids_weight[i]
          float alpha = 1.0 / h_centroids_weight[i];
          status = hipblasSscal (
              handle, partitions.n_dimensions, &alpha,
              partitions.d_new_centroids + i * partitions.n_dimensions, 1);
          // check for errors
          CHECK_ERROR_RUNTIME_ERROR (
              status == HIPBLAS_STATUS_SUCCESS,
              {
                hipblasDestroy (handle);
                free (h_centroids_weight);
              },
              "CUBLAS scal failed with status " + std::to_string (status));
        }
    }

  // syncronize threads
  err = hipDeviceSynchronize ();
  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (
      err == hipSuccess,
      {
        hipblasDestroy (handle);
        free (h_centroids_weight);
      },
      hipGetErrorString (err));

  // free data
  status = hipblasDestroy (handle);
  free (h_centroids_weight);

  return energy;
}

__host__ std::vector<float>
cudafcm (const std::vector<float> &data, const std::vector<float> &weights,
         const std::vector<float> &centroids, size_t n_dimensions,
         float tollerance, std::ofstream &log_stream)
{
  // check gpu properties
  int device_count;
  int device;
  hipError_t err;
  hipDeviceProp_t prop;

  // Inizializza CUDA
  err = hipGetDeviceCount (&device_count);
  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, PASS_INSTRUCTION,
                             hipGetErrorString (err));
  // check if there are devices
  CHECK_ERROR_RUNTIME_ERROR (device_count > 0, PASS_INSTRUCTION,
                             "No CUDA devices found");
  err = hipSetDevice (0);
  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, PASS_INSTRUCTION,
                             hipGetErrorString (err));
  err = hipGetDevice (&device);
  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, PASS_INSTRUCTION,
                             hipGetErrorString (err));

  // get gpu properties
  err = hipGetDeviceProperties (&prop, device);

  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, PASS_INSTRUCTION,
                             hipGetErrorString (err));
  // check if num of centroids is less then threads per block
  CHECK_ERROR_RUNTIME_ERROR (
      centroids.size () / n_dimensions <= prop.maxThreadsPerBlock,
      PASS_INSTRUCTION,
      "Number of centroids is greater than threads per block");
  // check if max num of threads per block is correct
  CHECK_ERROR_RUNTIME_ERROR (
      prop.maxThreadsPerBlock >= MAX_THREADS_PER_BLOCK, PASS_INSTRUCTION,
      "Number of threads per block is less than required");
  // check if shared memory is enough
  CHECK_ERROR_RUNTIME_ERROR (
      prop.sharedMemPerBlock >= MAX_THREADS_PER_BLOCK * sizeof (float),
      PASS_INSTRUCTION, "Shared memory is less than required");

  // report on the log file
  LOGGER (log_stream, "INFO", "Using device: " + std::string (prop.name));
  LOGGER (log_stream, "DEBUG",
          "Size of data points: " + std::to_string (n_dimensions));
  LOGGER (log_stream, "DEBUG", "Tollerance: " + std::to_string (tollerance));
  LOGGER (log_stream, "DEBUG",
          "Number of centroids: "
              + std::to_string (centroids.size () / n_dimensions));
  LOGGER (log_stream, "DEBUG",
          "Number of data points: "
              + std::to_string (data.size () / n_dimensions));

  // check if number of centroids is supported
  CHECK_ERROR_RUNTIME_ERROR(
      centroids.size () / n_dimensions <= MAX_THREADS_PER_BLOCK,
      PASS_INSTRUCTION, "Number of centroids not supported");

  // prepare cuda context:
  size_t batch_size = 0; // number of data points to use in a single batch
  float *d_main_ptr;     // pointer to the main memory pool
  {
    // reserved memory for system
    size_t reserved_memory
        = (float)(GiB_SYS_RESERVED_MEM) * 1024 * 1024 * 1024;
    // set a memory pool for the device over the remaining memory
    size_t free, total;
    hipMemGetInfo (&free, &total);
    size_t pool_memory = free - reserved_memory;
    LOGGER (log_stream, "INFO",
            "Total memory: " + std::to_string (total) + " bytes");
    LOGGER (log_stream, "INFO",
            "Free memory: " + std::to_string (free) + " bytes");
    /**
     * @remark
     * C is number of centroids
     * D is number of dimensions
     * N is number of data points
     * The required memory is:
     *   2 x C x D + N x D + 2 x N + N x C
     * = 2 x C x D + (D + 2 + C) x N
     * The number of used data points is:
     *   (pool_memory - 2 x C x D) / (D + 2 + C)
     */

    // compute the number of data points to use
    batch_size = (pool_memory - 2 * centroids.size () * sizeof (float))
                 / ((n_dimensions + centroids.size () / n_dimensions + 2)
                    * sizeof (float));
    batch_size = std::min (batch_size, data.size () / n_dimensions);

    // check if batch_size is less than 0
    CHECK_ERROR_BAD_ALLOC (batch_size > 0, PASS_INSTRUCTION,
                           "Not enough memory to allocate the data points");
    // check if batch_size is greater than max number of activable
    // blocks
    CHECK_ERROR_BAD_ALLOC (
        batch_size <= prop.maxGridSize[0], PASS_INSTRUCTION,
        "Number of data points is greater than max number of "
        "activable blocks");

    LOGGER (log_stream, "INFO", "batch_size: " + std::to_string (batch_size));

    // compute the total memory required
    size_t total_memory
        = (2 * centroids.size ()
           + batch_size
                 * (n_dimensions + 2 + centroids.size () / n_dimensions))
          * sizeof (float);

    // try to allocate memory
    err = hipMalloc (&d_main_ptr, total_memory);
    // check for errors
    CHECK_ERROR_BAD_ALLOC (err == hipSuccess, PASS_INSTRUCTION,
                           hipGetErrorString (err));
  }

  // prepare data
  struct partition partitions = {
    .d_new_centroids = d_main_ptr,                 // len = centroids.size()
    .d_centroids = d_main_ptr + centroids.size (), // len = centroids.size()
    .d_data
    = d_main_ptr + 2 * centroids.size (), // len = batch_size * n_dimensions
    .d_weights = d_main_ptr + 2 * centroids.size ()
                 + batch_size * n_dimensions, // len = batch_size
    .d_matrix = d_main_ptr + 2 * centroids.size () + batch_size * n_dimensions
                + batch_size, // len = batch_size * centroids.size() /
                              // n_dimensions
    .d_energies = d_main_ptr + 2 * centroids.size () + batch_size * n_dimensions
                  + batch_size + batch_size * centroids.size ()
                      / n_dimensions, // len = batch_size
    .batch_size = batch_size,
    .n_centroids = centroids.size () / n_dimensions,
    .n_dimensions = n_dimensions,
  };

  // copy centroids to the device
  err = hipMemcpy (partitions.d_centroids, centroids.data (),
                    centroids.size () * sizeof (float),
                    hipMemcpyHostToDevice);
  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, hipFree (d_main_ptr),
                             hipGetErrorString (err));

  // update centroids
  try
    {
      float pred_energy;                    // energy of the previous iteration
      float energy = INFINITY;              // current energy
      do
        {
          pred_energy = energy;  // safe energy in pred_energy

          // update centroids
          energy = compute_centroids (data, weights, partitions, prop,
                                            log_stream);
          LOGGER (log_stream, "INFO",
                  "energy: " + std::to_string (energy));

          // move d_new_centroids to d_centroids
          err = hipMemcpy (partitions.d_centroids, partitions.d_new_centroids,
                            centroids.size () * sizeof (float),
                            hipMemcpyDeviceToDevice);
          // check for errors
          CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, hipFree (d_main_ptr),
                                     hipGetErrorString (err));

          // check stability
          if (pred_energy < energy) {
            LOGGER(log_stream, "WARNING", "Numerical stability error");
          }
        }
      while (energy >= tollerance && pred_energy - energy >= tollerance);
    }
  catch (std::bad_alloc &e)
    {
      LOGGER (log_stream, "CRITICAL caught", e.what ());
      hipFree (d_main_ptr);
      throw std::runtime_error (e.what ());
    }
  catch (std::runtime_error &e)
    {
      LOGGER (log_stream, "CRITICAL caught", e.what ());
      hipFree (d_main_ptr);
      throw std::runtime_error (e.what ());
    }

  std::vector<float> out_centroids;

  // copy centroids from the device
  out_centroids.resize (centroids.size ());
  err = hipMemcpy (out_centroids.data (), partitions.d_centroids,
                    centroids.size () * sizeof (float),
                    hipMemcpyDeviceToHost);
  // check for errors
  CHECK_ERROR_RUNTIME_ERROR (err == hipSuccess, hipFree (d_main_ptr),
                             hipGetErrorString (err));

  // free memory
  hipFree (d_main_ptr);

  return out_centroids;
}
